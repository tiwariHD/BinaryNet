#include "hip/hip_runtime.h"
// This uses a lot of code from Caffe (http://caffe.berkeleyvision.org/);
// sources are clearly marked. Below we reproduce the original license of
// the Caffe software.
/*
Copyright (c) 2014, The Regents of the University of California (Regents)
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#undef _GLIBCXX_ATOMIC_BUILTINS


#define DIM_X  16
#define DIM_Y  16

// =============================================================================
// A x B
// size of work for a thread block
#define BLK_M_nn  96
#define BLK_N_nn  96

#define BLK_K  16

// size of thread block for reading A (dev->regs->shmem)
#define DIM_XA  32
#define DIM_YA  8

// size of thread block for reading B (dev->regs->shmem)
#define DIM_XB  8
#define DIM_YB  32

// =============================================================================
#define BLK_M BLK_M_nn
#define BLK_N BLK_N_nn
// =============================================================================

// size of work for a thread
#define THR_M ( BLK_M / DIM_X )
#define THR_N ( BLK_N / DIM_Y )

/******************************************************************************/

#define min(a, b) ((a) < (b) ? (a) : (b))

#define CEIL(x) (((x) + 1) / 1)


// (borrowed from Caffe: https://github.com/BVLC/caffe/blob/master/src/caffe/caffe_common.hpp)
// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n)                        \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n);                                       \
       i += blockDim.x * gridDim.x)

// CUDA: thread number configuration.
// Use 1024 threads per block, which requires cuda sm_2x or above,
// or fall back to attempt compatibility (best of luck to you).
#if __CUDA_ARCH__ >= 200
    const int CUDA_NUM_THREADS = 1024;
#else
    const int CUDA_NUM_THREADS = 512;
#endif

// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}


// (borrowed from Caffe: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu)
// Kernels for fast unfold + copy
// CUDA kernel for the case of dilation
__global__ void dilated_im2col_kernel(const int n, const float* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    float* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    float* data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const float* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        *data_col_ptr =
          (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
            data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

__global__ void im2col_kernel(const int n, const float* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    float* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    float* data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const float* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i ;
        int w_im = w_offset + j ;
        *data_col_ptr =
          (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
           data_im_ptr[i * width + j] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

void im2col(const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    float* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int dil_kernel_h = (kernel_h - 1) * dilation_h + 1;
  int dil_kernel_w = (kernel_w - 1) * dilation_w + 1;
  int height_col = (height + 2 * pad_h - dil_kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - dil_kernel_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  if(dilation_h != 1 || dilation_w != 1){
    dilated_im2col_kernel<<<GET_BLOCKS(num_kernels),
                  CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w,
      dilation_h, dilation_w, pad_h, pad_w, stride_h, stride_w, height_col,
      width_col, data_col);
  }
  else{
    im2col_kernel<<<GET_BLOCKS(num_kernels),
                  CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w,
      pad_h, pad_w, stride_h, stride_w, height_col,
      width_col, data_col);
  }
}

__global__ void im2col_gpu_int_kernel(const int n, const unsigned int* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    unsigned int* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    unsigned int* data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const unsigned int* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i ;
        int w_im = w_offset + j ;
        *data_col_ptr =
          (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
           data_im_ptr[i * width + j] : 0x55555555;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

void im2col_gpu_int(const unsigned int* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    unsigned int* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int dil_kernel_h = (kernel_h - 1) * dilation_h + 1;
  int dil_kernel_w = (kernel_w - 1) * dilation_w + 1;
  int height_col = (height + 2 * pad_h - dil_kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - dil_kernel_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  if(dilation_h != 1 || dilation_w != 1){
     PyErr_Format(PyExc_RuntimeError, "LUL ho gya\n");
    
  }
  else{
    im2col_gpu_int_kernel<<<GET_BLOCKS(num_kernels),
                  CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w,
      pad_h, pad_w, stride_h, stride_w, height_col,
      width_col, data_col);
  }
}

__global__ void my_xnor_gemm_kernel(
    int M, int N, int K,
    const unsigned int* __restrict__ A, int LDA,
    const unsigned int* __restrict__ B, int LDB,
    float*       __restrict__ C, int LDC,
    int offsetA, int offsetB )
{

    int idx = threadIdx.x;  // thread's m dimension
    int idy = threadIdx.y;  // thread's n dimension

    int idt = DIM_X * idy + idx;    // thread's global number

    int idxA = idt % DIM_XA;    // idx within A
    int idyA = idt / DIM_XA;    // idy within A

    int idxB = idt % DIM_XB;    // idx within B
    int idyB = idt / DIM_XB;    // idy within B

    int blx = blockIdx.x;   // block's m dimension
    int bly = blockIdx.y;   // block's n dimension

    __shared__ unsigned int sA[BLK_K][BLK_M+1];      // +1 only required if A is transposed
    __shared__ unsigned int sB[BLK_N][BLK_K+1];      // +1 always required

    // Registers for the innermost loop
    unsigned int rC[THR_N][THR_M];
    unsigned int rA[THR_M];
    unsigned int rB[THR_N];

    // Registers for the dev->shmem copy
    unsigned int ra[BLK_K/DIM_YA][BLK_M/DIM_XA];
    unsigned int rb[BLK_N/DIM_YB][BLK_K/DIM_XB];

    const unsigned int  *offs_dA = A + blx*BLK_M     + idyA*LDA + idxA;
    ptrdiff_t boundA = (LDA*(K-1) + M) - (blx*BLK_M + idyA*LDA + idxA) - 1;
    const unsigned int *offs_dB = B + bly*BLK_N*LDB + idyB*LDB + idxB;
    ptrdiff_t boundB = (LDB*(N-1) + K) - ( bly*BLK_N*LDB + idyB*LDB + idxB ) -1;

    int m, n, k, kk;

    // Zero C
    #pragma unroll
    for (n = 0; n < THR_N; n++)
        #pragma unroll
        for (m = 0; m < THR_M; m++)
            rC[n][m] = 0;

    // Load A dev->shmem
        #pragma unroll
        for (n = 0; n < BLK_K; n += DIM_YA)
            #pragma unroll
            for (m = 0; m < BLK_M; m += DIM_XA)
                sA[n+idyA][m+idxA] = offs_dA[min(n*LDA+m, boundA)];

    // Load B dev->shmem
        #pragma unroll
        for (n = 0; n < BLK_N; n += DIM_YB)
            #pragma unroll
            for (m = 0; m < BLK_K; m += DIM_XB)
                sB[n+idyB][m+idxB] = offs_dB[min(n*LDB+m, boundB)];

    __syncthreads();

    for (kk = 0; kk < K-BLK_K; kk += BLK_K)
    {
        offs_dA += BLK_K*LDA;
        boundA  -= BLK_K*LDA;
        offs_dB += BLK_K;
        boundB  -= BLK_K;

        // Load A dev->regs
            #pragma unroll
            for (n = 0; n < BLK_K/DIM_YA; n++)
                #pragma unroll
                for (m = 0; m < BLK_M/DIM_XA; m++)
                    ra[n][m] = offs_dA[min(n*DIM_YA*LDA + m*DIM_XA, boundA)];

        // Load B dev->regs
            #pragma unroll
            for (n = 0; n < BLK_N/DIM_YB; n++)
                #pragma unroll
                for (m = 0; m < BLK_K/DIM_XB; m++)
                    rb[n][m] = offs_dB[min(n*DIM_YB*LDB + m*DIM_XB, boundB)];

        // Multiply
        #pragma unroll
        for (k = 0; k < BLK_K; k++)
        {
            // Load A shmem->regs
            #pragma unroll
            for (m = 0; m < THR_M; m++)
                rA[m] = sA[k][m*DIM_X+idx];

            // Load B shmem->regs
            #pragma unroll
            for (n = 0; n < THR_N; n++)
                rB[n] = sB[n*DIM_Y+idy][k];

            // Compute
            #pragma unroll
            for (n = 0; n < THR_N; n++) {
                #pragma unroll
                for (m = 0; m < THR_M; m++) {
                        rC[n][m] += __popc(rA[m] ^ rB[n]);
                }
            }
        }

        __syncthreads();

        // Load A regs->shmem
            #pragma unroll
            for (n = 0; n < BLK_K/DIM_YA; n++)
                #pragma unroll
                for (m = 0; m < BLK_M/DIM_XA; m++)
                    sA[n*DIM_YA+idyA][m*DIM_XA+idxA] = ra[n][m];

        // Load B regs->shmem
            #pragma unroll
            for (n = 0; n < BLK_N/DIM_YB; n++)
                #pragma unroll
                for (m = 0; m < BLK_K/DIM_XB; m++)
                    sB[n*DIM_YB+idyB][m*DIM_XB+idxB] = rb[n][m];

        __syncthreads();
    }

    // Multiply last full (BLK_K) or partial block of
    // columns of op(A) and rows of op(B).
    // It's okay that m,n exceed matrix bounds as all work is in registers
    // or shared memory, and out-of-bounds rC[n][m] will not be saved later.
    kk = K - kk;
    #pragma unroll
    for (k = 0; k < kk; k++)
    {
        // Load A shmem->regs
        #pragma unroll
        for (m = 0; m < THR_M; m++)
            rA[m] = sA[k][m*DIM_X+idx];

        // Load B shmem->regs
        #pragma unroll
        for (n = 0; n < THR_N; n++)
            rB[n] = sB[n*DIM_Y+idy][k];

        // Compute
        #pragma unroll
        for (n = 0; n < THR_N; n++) {
            #pragma unroll
            for (m = 0; m < THR_M; m++) {
                    rC[n][m] += __popc(rA[m] ^ rB[n]);
            }
        }
    }

    // Store C regs->dev
    #pragma unroll
    for (n = 0; n < THR_N; n++) {
        int coord_dCn = bly*BLK_N + n*DIM_Y + idy;
        #pragma unroll
        for (m = 0; m < THR_M; m++) {
            int coord_dCm = blx*BLK_M + m*DIM_X + idx;
            if (coord_dCm < M && coord_dCn < N) {
                int offsC = coord_dCn*LDC + coord_dCm;

                unsigned int &regC = rC[n][m];
                float &memC = C[offsC];

		memC = -((2 * (float)regC) - (32 * K));
            }
        }
    }
}


__device__ unsigned int concatenate(float* array)
{
    unsigned int rvalue=0;
    unsigned int sign;
            
    for (int i = 0; i < 32; i++)
    {
       sign = (array[i]>0);
       rvalue = rvalue | (sign<<i);
    }
                                   
    return rvalue;
}

__global__ void concatenate_input_kernel(float *a, unsigned int *b, int height, int width)
{   
    int size = height * width;
    int out_stride = blockIdx.x * size;
    int in_stride = 32 * out_stride;
    float* array = new float[32];    

    for(int j = threadIdx.x; j < size; j += blockDim.x) {
        for(int k = 0; k < 32; k++)
            array[k] = a[j + in_stride + (k*size)];
        b[j + out_stride] = concatenate(array); 
    }
    delete[] array;
}



// CUDA kernel for the case of dilation
__global__ void dilated_col2im_kernel(const int n, const float* data_col,
    const int height, const int width, const int channels,
    const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    float* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    float val = 0;
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int c_im = index / (width * height);
    int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
    // compute the start and end of the output
    const int w_col_start =
        (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
      for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
        int h_k = (h_im - h_col * stride_h);
        int w_k = (w_im - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int data_col_index = (((c_im * kernel_h + h_k) * kernel_w + w_k) *
                                height_col + h_col) * width_col + w_col;
          val += data_col[data_col_index];
        }
      }
    }
    data_im[index] = val;
  }
}

__global__ void col2im_kernel(const int n, const float* data_col,
    const int height, const int width, const int channels,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    float* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    float val = 0;
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int c_im = index / (width * height);
    // compute the start and end of the output
    const int w_col_start =
        (w_im < kernel_w) ? 0 : (w_im - kernel_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_h) ? 0 : (h_im - kernel_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    // equivalent implementation, no dilation
    int offset =
      (c_im * kernel_h * kernel_w + h_im * kernel_w + w_im) * height_col * width_col;
    int coeff_h_col = (1 - stride_h * kernel_w * height_col) * width_col;
    int coeff_w_col = (1 - stride_w * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

void col2im(const float* data_col, const int channels,
    const int height, const int width, const int patch_h, const int patch_w,
    const int dilation_h, const int dilation_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, float* data_im) {
  int dil_patch_h = (patch_h - 1) * dilation_h + 1;
  int dil_patch_w = (patch_w - 1) * dilation_w + 1;
  int height_col = (height + 2 * pad_h - dil_patch_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - dil_patch_w) / stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  if(dilation_h != 1 || dilation_w != 1){
    dilated_col2im_kernel<<<GET_BLOCKS(num_kernels),
                  CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, channels, patch_h, patch_w,
      dilation_h, dilation_w, pad_h, pad_w, stride_h, stride_w,
      height_col, width_col, data_im);
  }
  else{
    col2im_kernel<<<GET_BLOCKS(num_kernels),
                  CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, channels, patch_h, patch_w,
      pad_h, pad_w, stride_h, stride_w,
      height_col, width_col, data_im);
  }
}

// Theano op code
// Authors: Arjun Jain, Frederic Bastien, Jan Schluter
// Reference code: https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu
//   and https://github.com/torch/cunn/blob/master/SpatialConvolutionMM.cu
CudaNdarray* corrMM(CudaNdarray *const bottom,
                    CudaNdarray *const weight,
                    CudaNdarray *const top,
                    const int direction,
                    const int dH = 1,
                    const int dW = 1,
                    const int dilH = 1,
                    const int dilW = 1,
                    const int padH = 0,
                    const int padW = 0)
{
    if (bottom->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "GpuCorrMM requires bottom of 4D");
        return NULL;
    }
    if (!CudaNdarray_is_c_contiguous(bottom))
    {
        PyErr_Format(PyExc_ValueError,
                "GpuCorrMM requires bottom to be C-contiguous, "
                "but strides are: %d %d %d %d\n",
                CudaNdarray_HOST_STRIDES(bottom)[0],
                CudaNdarray_HOST_STRIDES(bottom)[1],
                CudaNdarray_HOST_STRIDES(bottom)[2],
                CudaNdarray_HOST_STRIDES(bottom)[3]);
        return NULL;
    }

    if (weight->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "GpuCorrMM requires weight of 4D");
        return NULL;
    }
    if (!CudaNdarray_is_c_contiguous(weight))
    {
        PyErr_Format(PyExc_ValueError,
                "GpuCorrMM requires weight to be C-contiguous, "
                "but strides are: %d %d %d %d\n",
                CudaNdarray_HOST_STRIDES(weight)[0],
                CudaNdarray_HOST_STRIDES(weight)[1],
                CudaNdarray_HOST_STRIDES(weight)[2],
                CudaNdarray_HOST_STRIDES(weight)[3]);
        return NULL;
    }

    if (top->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "GpuCorrMM requires top of 4D");
        return NULL;
    }
    if (!CudaNdarray_is_c_contiguous(top))
    {
        PyErr_Format(PyExc_ValueError,
                "GpuCorrMM requires top to be C-contiguous, "
                "but strides are: %d %d %d %d\n",
                CudaNdarray_HOST_STRIDES(top)[0],
                CudaNdarray_HOST_STRIDES(top)[1],
                CudaNdarray_HOST_STRIDES(top)[2],
                CudaNdarray_HOST_STRIDES(top)[3]);
        return NULL;
    }

    // Extract some shape information for later and check shape consistency
    // bottom: (batchSize, nChannels, bottomHeight, bottomWidth)
    const int batchSize = CudaNdarray_HOST_DIMS(bottom)[0];
    const int nChannels = CudaNdarray_HOST_DIMS(bottom)[1];
    const int bottomHeight = CudaNdarray_HOST_DIMS(bottom)[2];
    const int bottomWidth = CudaNdarray_HOST_DIMS(bottom)[3];
    // weights: (nFilters, nChannels, rows, columns)
    const int nFilters = CudaNdarray_HOST_DIMS(weight)[0];
    const int kH = CudaNdarray_HOST_DIMS(weight)[2];
    const int kW = CudaNdarray_HOST_DIMS(weight)[3];
    if (nChannels != CudaNdarray_HOST_DIMS(weight)[1]) {
        PyErr_SetString(PyExc_ValueError,
                "GpuCorrMM images and kernel must have the same stack size\n");
        return NULL;
    }
    // implicit dilated filter
    const int dil_kH = (kH - 1) * dilH + 1;
    const int dil_kW = (kW - 1) * dilW + 1;
    // top: (batchSize, nFilters, topHeight, topWidth)
    const int topHeightNoDH = (bottomHeight + 2*padH - dil_kH);
    const int topWidthNoDW  = (bottomWidth + 2*padW - dil_kW);
    // the above values might be negative so we need to use Python-like
    // flooring integer division to be compatible with get_conv_output.
    // note: this macro implements Python's // for negative x only
#define _CONV_FLOORDIV_X(x,y) ((x < 0) ? (- ((-x) / y) - (((-x) % y) == 0 ? 0 : 1)) : (x / y))
    const int topHeight = _CONV_FLOORDIV_X(topHeightNoDH, dH) + 1;
    const int topWidth  = _CONV_FLOORDIV_X(topWidthNoDW, dW) + 1;
#undef _CONV_FLOORDIV
    if (batchSize != CudaNdarray_HOST_DIMS(top)[0] ||
            nFilters != CudaNdarray_HOST_DIMS(top)[1] ||
            topHeight != CudaNdarray_HOST_DIMS(top)[2] ||
            topWidth != CudaNdarray_HOST_DIMS(top)[3]) {
        PyErr_Format(PyExc_ValueError,
                "GpuCorrMM shape inconsistency:\n"
                "  bottom shape: %d %d %d %d\n"
                "  weight shape: %d %d %d %d\n"
                "  top shape: %d %d %d %d (expected %d %d %d %d)\n",
                batchSize, nChannels, bottomHeight, bottomWidth,
                nFilters, nChannels, kH, kW,
                CudaNdarray_HOST_DIMS(top)[0], CudaNdarray_HOST_DIMS(top)[1],
                CudaNdarray_HOST_DIMS(top)[2], CudaNdarray_HOST_DIMS(top)[3],
                batchSize, nFilters, topHeight, topWidth);
        return NULL;
    }

    // Create temporary columns
    int col_dim[2];
    col_dim[0] = nChannels * kW * kH;
    col_dim[1] = topHeight * topWidth;
    CudaNdarray* col = (CudaNdarray*)CudaNdarray_NewDims(2, col_dim);
    if (NULL == col)
    {
        PyErr_Format(PyExc_RuntimeError,
                "GpuCorrMM failed to allocate working memory of %d x %d\n",
                col_dim[0], col_dim[1]);
        return NULL;
    }

    // Define some useful variables
    const int bottom_stride = CudaNdarray_HOST_STRIDES(bottom)[0];
    const int top_stride = CudaNdarray_HOST_STRIDES(top)[0];
    const int K_ = col_dim[0];
    const int N_ = col_dim[1];
    const int M_ = nFilters;
    const float one = 1.0f;
    const float zero = 0.0f;

    CudaNdarray *output;
    if (direction == 0) {  // forward pass
        output = top;
        if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
            hipError_t err = hipMemset(output->devdata, 0,
                                         CudaNdarray_SIZE(output) * sizeof(real));
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM could not fill the output with zeros: %s",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            Py_DECREF(col);
            return output;
        }
        // valid correlation: im2col, then gemm
        // Iterate over batch
        for (int n = 0; n < batchSize; n++) {
            // First, im2col
            im2col(bottom->devdata + n * bottom_stride, nChannels, bottomHeight,
                   bottomWidth, kH, kW, dilH, dilW,
                   padH, padW, dH, dW, col->devdata);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM encountered a CUDA error in im2col: %s\n"
                             "This could be a known bug in CUDA, please see the "
                             "GpuCorrMM() documentation.\n",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            // Second, gemm
            hipblasStatus_t status = hipblasSgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N_, M_, K_,
                    &one,
                    col->devdata, N_,
                    weight->devdata, K_,
                    &zero,
                    top->devdata + n * top_stride, N_);
            if (status != HIPBLAS_STATUS_SUCCESS) {
                PyErr_Format(PyExc_RuntimeError,
                        "GpuCorrMM encountered a CUBLAS error: %s\n"
                        "This could be a known bug in CUDA, please see the "
                        "GpuCorrMM() documentation.\n",
                        cublasGetErrorString(status));
                Py_DECREF(col);
                return NULL;
            }
        }
        /*
        // Original caffe code for comparison
        // https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu
        // Note that this is for grouped convolution; we can ignore groups here,
        // but the group-related offsets help explain what M_, N_ and K_ are
        int weight_offset = M_ * K_;
        int col_offset = K_ * N_;
        int top_offset = M_ * N_;
        for (int n = 0; n < num_; ++n) {
          // First, im2col
          im2col_gpu(bottom_data + bottom[i]->offset(n), channels_, height_,
              width_, kernel_h_, kernel_w_, pad_h_, pad_w_, stride_h_, stride_w_,
              col_data);
          // Second, innerproduct with groups
          for (int g = 0; g < group_; ++g) {
            caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, K_,
              (Dtype)1., weight + weight_offset * g, col_data + col_offset * g,
              (Dtype)0., top_data + (*top)[i]->offset(n) + top_offset * g);
            == (see https://github.com/BVLC/caffe/blob/master/src/caffe/util/math_functions.cu#L16)
            hipblasSgemm(HIPBLAS_OP_N, HIPBLAS_OP_N,
              N_, M_, K_,
              1.,
              col_data + col_offset * g, N_,
              weight + weight_offset * g, K_,
              0.,
              top_data + (*top)[i]->offset(n) + top_offset * g, N_);
          }
        }
        */
    }
    else if (direction == 1) {  // backprop wrt. weights
        output = weight;
        if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
            hipError_t err = hipMemset(output->devdata, 0,
                                         CudaNdarray_SIZE(output) * sizeof(real));
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM grad wrt. weights could not fill the output with zeros: %s",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            Py_DECREF(col);
            return output;
        }
        // valid convolution: im2col, then gemm
        // Iterate over batch
        for (int n = 0; n < batchSize; n++) {
            // First, im2col
            im2col(bottom->devdata + n * bottom_stride, nChannels, bottomHeight,
                   bottomWidth, kH, kW, dilH, dilW,
                   padH, padW, dH, dW, col->devdata);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM encountered a CUDA error in im2col: %s\n"
                             "This could be a known bug in CUDA, please see the "
                             "GpuCorrMM() documentation.\n",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            // Second, gemm
            // Note that we accumulate into weight. We do so by setting beta = 0
            // for the first iteration and beta = 1 for subsequent ones. (This
            // is faster than setting weight to all zeros before the loop.)
            hipblasStatus_t status = hipblasSgemm(handle,
                    HIPBLAS_OP_T, HIPBLAS_OP_N,
                    K_, M_, N_,
                    &one,
                    col->devdata, N_,
                    top->devdata + n * top_stride, N_,
                    (n == 0) ? &zero : &one,
                    weight->devdata, K_);
            if (status != HIPBLAS_STATUS_SUCCESS) {
                PyErr_Format(PyExc_RuntimeError,
                        "GpuCorrMM encountered a CUBLAS error: %s\n"
                        "This could be a known bug in CUDA, please see the "
                        "GpuCorrMM() documentation.\n",
                        cublasGetErrorString(status));
                Py_DECREF(col);
                return NULL;
            }
        }
        /*
        // Original caffe code for comparison
        // https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu
        // Note that this is for grouped convolution; we can ignore groups
        for (int n = 0; n < num_; ++n) {
          // Since we saved memory in the forward pass by not storing all col
          // data, we will need to recompute them.
          im2col_gpu(bottom_data + (*bottom)[i]->offset(n), channels_, height_,
                     width_, kernel_h_, kernel_w_, pad_h_, pad_w_,
                     stride_h_, stride_w_, col_data);
          // gradient w.r.t. weight. Note that we will accumulate diffs.
          for (int g = 0; g < group_; ++g) {
            caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, K_, N_,
                (Dtype)1., top_diff + top[i]->offset(n) + top_offset * g,
                col_data + col_offset * g, (Dtype)1.,
                weight_diff + weight_offset * g);
            == (see https://github.com/BVLC/caffe/blob/master/src/caffe/util/math_functions.cu#L16)
            hipblasSgemm(HIPBLAS_OP_T, HIPBLAS_OP_N, K_, M_, N_,
                1.0,
                col_data + col_offset * g, N_,
                top_diff + top[i]->offset(n) + top_offset * g, N_,
                1.0,
                weight_diff + weight_offset * g, K_);
          }
        }
        */
    }
    else if (direction == 2) {  // backprop wrt. inputs
        output = bottom;
        if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
            hipError_t err = hipMemset(output->devdata, 0,
                                         CudaNdarray_SIZE(output) * sizeof(real));
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM grad wrt. inputs could not fill the output with zeros: %s",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            Py_DECREF(col);
            return output;
        }
        // full convolution: gemm, then col2im
        // Iterate over batch
        for (int n = 0; n < batchSize; n++) {
            // gemm into columns
            hipblasStatus_t status = hipblasSgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_T,
                    N_, K_, M_,
                    &one,
                    top->devdata + n * top_stride, N_,
                    weight->devdata, K_,
                    &zero,
                    col->devdata, N_);
            if (status != HIPBLAS_STATUS_SUCCESS) {
                PyErr_Format(PyExc_RuntimeError,
                        "GpuCorrMM encountered a CUBLAS error: %s\n"
                        "This could be a known bug in CUDA, please see the "
                        "GpuCorrMM() documentation.\n",
                        cublasGetErrorString(status));
                Py_DECREF(col);
                return NULL;
            }
            // col2im back to the data
            col2im(col->devdata, nChannels, bottomHeight, bottomWidth,
                   kH, kW, dilH, dilW, padH, padW,
                   dH, dW, bottom->devdata + n * bottom_stride);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM encountered a CUDA error in col2im: %s\n"
                             "This could be a known bug in CUDA, please see the "
                             "GpuCorrMM() documentation.\n",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
        }
        /*
        // Original caffe code for comparison
        // https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu
        for (int n = 0; n < num_; ++n) {
          // gradient w.r.t. bottom data, if necessary
          if (propagate_down[i]) {
            for (int g = 0; g < group_; ++g) {
              caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, K_, N_, M_,
                  (Dtype)1., weight + weight_offset * g,
                  top_diff + top[i]->offset(n) + top_offset * g,
                  (Dtype)0., col_diff + col_offset * g);
              == (see https://github.com/BVLC/caffe/blob/master/src/caffe/util/math_functions.cu#L16)
              hipblasSgemm(HIPBLAS_OP_N, HIPBLAS_OP_T, N_, K_, M_,
                  1.,
                  top_diff + top[i]->offset(n) + top_offset * g, N_,
                  weight + weight_offset * g, K_,
                  0.,
                  col_diff + col_offset * g, N_);
            }
            // col2im back to the data
            col2im_gpu(col_diff, channels_, height_, width_,
                kernel_h_, kernel_w_, pad_h_, pad_w_, stride_h_, stride_w_,
                bottom_diff + (*bottom)[i]->offset(n));
          }
        }
        */
    }
    // Free temporary columns
    Py_DECREF(col);

    // Note that we don't change the refcount of the output matrix here. Output
    // (re)allocation and refcounting is done in BaseGpuCorrMM.c_code_helper();
    // in here output is just aliased to one of bottom, weights, or top.
    return output;
}

CudaNdarray* corrMMBinary(CudaNdarray *const bottom,
                    CudaNdarray *const weight,
                    CudaNdarray *const top,
                    const int direction,
                    const int dH = 1,
                    const int dW = 1,
                    const int dilH = 1,
                    const int dilW = 1,
                    const int padH = 0,
                    const int padW = 0)
{
    if (bottom->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "GpuCorrMM requires bottom of 4D");
        return NULL;
    }
    if (!CudaNdarray_is_c_contiguous(bottom))
    {
        PyErr_Format(PyExc_ValueError,
                "GpuCorrMM requires bottom to be C-contiguous, "
                "but strides are: %d %d %d %d\n",
                CudaNdarray_HOST_STRIDES(bottom)[0],
                CudaNdarray_HOST_STRIDES(bottom)[1],
                CudaNdarray_HOST_STRIDES(bottom)[2],
                CudaNdarray_HOST_STRIDES(bottom)[3]);
        return NULL;
    }

    if (weight->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "GpuCorrMM requires weight of 4D");
        return NULL;
    }
    if (!CudaNdarray_is_c_contiguous(weight))
    {
        PyErr_Format(PyExc_ValueError,
                "GpuCorrMM requires weight to be C-contiguous, "
                "but strides are: %d %d %d %d\n",
                CudaNdarray_HOST_STRIDES(weight)[0],
                CudaNdarray_HOST_STRIDES(weight)[1],
                CudaNdarray_HOST_STRIDES(weight)[2],
                CudaNdarray_HOST_STRIDES(weight)[3]);
        return NULL;
    }

    if (top->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "GpuCorrMM requires top of 4D");
        return NULL;
    }
    if (!CudaNdarray_is_c_contiguous(top))
    {
        PyErr_Format(PyExc_ValueError,
                "GpuCorrMM requires top to be C-contiguous, "
                "but strides are: %d %d %d %d\n",
                CudaNdarray_HOST_STRIDES(top)[0],
                CudaNdarray_HOST_STRIDES(top)[1],
                CudaNdarray_HOST_STRIDES(top)[2],
                CudaNdarray_HOST_STRIDES(top)[3]);
        return NULL;
    }

    // Extract some shape information for later and check shape consistency
    // bottom: (batchSize, nChannels, bottomHeight, bottomWidth)
    const int batchSize = CudaNdarray_HOST_DIMS(bottom)[0];
    const int nChannels = CudaNdarray_HOST_DIMS(bottom)[1];
    const int bottomHeight = CudaNdarray_HOST_DIMS(bottom)[2];
    const int bottomWidth = CudaNdarray_HOST_DIMS(bottom)[3];
    // weights: (nFilters, nChannels, rows, columns)
    const int nFilters = CudaNdarray_HOST_DIMS(weight)[0];
    const int kH = CudaNdarray_HOST_DIMS(weight)[2];
    const int kW = CudaNdarray_HOST_DIMS(weight)[3];
    if (nChannels != CudaNdarray_HOST_DIMS(weight)[1]) {
        PyErr_SetString(PyExc_ValueError,
                "GpuCorrMM images and kernel must have the same stack size\n");
        return NULL;
    }
    // implicit dilated filter
    const int dil_kH = (kH - 1) * dilH + 1;
    const int dil_kW = (kW - 1) * dilW + 1;
    // top: (batchSize, nFilters, topHeight, topWidth)
    const int topHeightNoDH = (bottomHeight + 2*padH - dil_kH);
    const int topWidthNoDW  = (bottomWidth + 2*padW - dil_kW);
    // the above values might be negative so we need to use Python-like
    // flooring integer division to be compatible with get_conv_output.
    // note: this macro implements Python's // for negative x only
#define _CONV_FLOORDIV_X(x,y) ((x < 0) ? (- ((-x) / y) - (((-x) % y) == 0 ? 0 : 1)) : (x / y))
    const int topHeight = _CONV_FLOORDIV_X(topHeightNoDH, dH) + 1;
    const int topWidth  = _CONV_FLOORDIV_X(topWidthNoDW, dW) + 1;
#undef _CONV_FLOORDIV
    if (batchSize != CudaNdarray_HOST_DIMS(top)[0] ||
            nFilters != CudaNdarray_HOST_DIMS(top)[1] ||
            topHeight != CudaNdarray_HOST_DIMS(top)[2] ||
            topWidth != CudaNdarray_HOST_DIMS(top)[3]) {
        PyErr_Format(PyExc_ValueError,
                "GpuCorrMM shape inconsistency:\n"
                "  bottom shape: %d %d %d %d\n"
                "  weight shape: %d %d %d %d\n"
                "  top shape: %d %d %d %d (expected %d %d %d %d)\n",
                batchSize, nChannels, bottomHeight, bottomWidth,
                nFilters, nChannels, kH, kW,
                CudaNdarray_HOST_DIMS(top)[0], CudaNdarray_HOST_DIMS(top)[1],
                CudaNdarray_HOST_DIMS(top)[2], CudaNdarray_HOST_DIMS(top)[3],
                batchSize, nFilters, topHeight, topWidth);
        return NULL;
    }

    // Create temporary columns
    int col_dim[2];
    col_dim[0] = nChannels * kW * kH;
    col_dim[1] = topHeight * topWidth;
    CudaNdarray* col = (CudaNdarray*)CudaNdarray_NewDims(2, col_dim);
    if (NULL == col)
    {
        PyErr_Format(PyExc_RuntimeError,
                "GpuCorrMM failed to allocate working memory of %d x %d\n",
                col_dim[0], col_dim[1]);
        return NULL;
    }


    // Define some useful variables
    const int bottom_stride = CudaNdarray_HOST_STRIDES(bottom)[0];
    const int top_stride = CudaNdarray_HOST_STRIDES(top)[0];
    const int K_ = col_dim[0];
    const int N_ = col_dim[1];
    const int M_ = nFilters;
    const float one = 1.0f;
    const float zero = 0.0f;

    CudaNdarray *output;
    if (direction == 0) {  // forward pass
        output = top;
        if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
            hipError_t err = hipMemset(output->devdata, 0,
                                         CudaNdarray_SIZE(output) * sizeof(real));
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM could not fill the output with zeros: %s",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            Py_DECREF(col);
            return output;
        }
  
        unsigned int* d_input_conc;
        unsigned int* d_kernel_conc;
        unsigned int* d_imColArr;
        d_input_conc = (unsigned int*)device_malloc(nChannels * bottomHeight * bottomWidth * sizeof(unsigned int) / 32);
        d_kernel_conc = (unsigned int*)device_malloc(nFilters * nChannels * kW * kH * sizeof(unsigned int) / 32);
        d_imColArr = (unsigned int*)device_malloc(topHeight * topWidth * nChannels * kW * kH * sizeof(unsigned int) / 32);

        // Concatenate weights
        concatenate_input_kernel<<< (nFilters * nChannels / 32), (kH * kW) >>>(weight->devdata, d_kernel_conc, kH, kW);

        // valid correlation: im2col, then gemm
        // Iterate over batch
        for (int n = 0; n < batchSize; n++) {
            // First, concatenate input
            concatenate_input_kernel<<< (nChannels / 32), min(bottomHeight, 256) >>>(
            bottom->devdata + n * bottom_stride, d_input_conc, bottomHeight, bottomWidth);

            // Second, im2col
            im2col_gpu_int(d_input_conc, nChannels / 32, bottomHeight, 
                bottomWidth, kH, kH, dilH, dilW, padH, padH, dH, dW, d_imColArr);
           
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM encountered a CUDA error in im2col: %s\n"
                             "This could be a known bug in CUDA, please see the "
                             "GpuCorrMM() documentation.\n",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }

            dim3 blockDim1(16, 16);
            int gridSize1 = CEIL((float)(N_) / (float)(96));
            int gridSize2 = CEIL((float)(M_) / (float)(96));
            dim3 gridDim1(gridSize1, gridSize2);
            my_xnor_gemm_kernel<<< gridDim1, blockDim1, 0 >>>(N_, M_, (K_)/32,
            d_imColArr, N_, d_kernel_conc, (K_)/32, top->devdata + n * top_stride, N_, 0, 0);
            err = hipGetLastError();
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                        "GpuCorrMM encountered a CUDA error in im2col: %s\n"
                        "This could be a known bug in CUDA, please see the "
                        "GpuCorrMM() documentation.\n",
                        hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
        }
        
        device_free(d_input_conc);
        d_input_conc = NULL;
        device_free(d_kernel_conc);
        d_kernel_conc = NULL;;
        device_free(d_imColArr);
        d_imColArr = NULL;;
    }
    else if (direction == 1) {  // backprop wrt. weights
        output = weight;
        if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
            hipError_t err = hipMemset(output->devdata, 0,
                                         CudaNdarray_SIZE(output) * sizeof(real));
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM grad wrt. weights could not fill the output with zeros: %s",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            Py_DECREF(col);
            return output;
        }
        // valid convolution: im2col, then gemm
        // Iterate over batch
        for (int n = 0; n < batchSize; n++) {
            // First, im2col
            im2col(bottom->devdata + n * bottom_stride, nChannels, bottomHeight,
                   bottomWidth, kH, kW, dilH, dilW,
                   padH, padW, dH, dW, col->devdata);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM encountered a CUDA error in im2col: %s\n"
                             "This could be a known bug in CUDA, please see the "
                             "GpuCorrMM() documentation.\n",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            // Second, gemm
            // Note that we accumulate into weight. We do so by setting beta = 0
            // for the first iteration and beta = 1 for subsequent ones. (This
            // is faster than setting weight to all zeros before the loop.)
            hipblasStatus_t status = hipblasSgemm(handle,
                    HIPBLAS_OP_T, HIPBLAS_OP_N,
                    K_, M_, N_,
                    &one,
                    col->devdata, N_,
                    top->devdata + n * top_stride, N_,
                    (n == 0) ? &zero : &one,
                    weight->devdata, K_);
            if (status != HIPBLAS_STATUS_SUCCESS) {
                PyErr_Format(PyExc_RuntimeError,
                        "GpuCorrMM encountered a CUBLAS error: %s\n"
                        "This could be a known bug in CUDA, please see the "
                        "GpuCorrMM() documentation.\n",
                        cublasGetErrorString(status));
                Py_DECREF(col);
                return NULL;
            }
        }
    }
    else if (direction == 2) {  // backprop wrt. inputs
        output = bottom;
        if (batchSize == 0 || nChannels == 0 || nFilters == 0) {
            hipError_t err = hipMemset(output->devdata, 0,
                                         CudaNdarray_SIZE(output) * sizeof(real));
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM grad wrt. inputs could not fill the output with zeros: %s",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
            Py_DECREF(col);
            return output;
        }
        // full convolution: gemm, then col2im
        // Iterate over batch
        for (int n = 0; n < batchSize; n++) {
            // gemm into columns
            hipblasStatus_t status = hipblasSgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_T,
                    N_, K_, M_,
                    &one,
                    top->devdata + n * top_stride, N_,
                    weight->devdata, K_,
                    &zero,
                    col->devdata, N_);
            if (status != HIPBLAS_STATUS_SUCCESS) {
                PyErr_Format(PyExc_RuntimeError,
                        "GpuCorrMM encountered a CUBLAS error: %s\n"
                        "This could be a known bug in CUDA, please see the "
                        "GpuCorrMM() documentation.\n",
                        cublasGetErrorString(status));
                Py_DECREF(col);
                return NULL;
            }
            // col2im back to the data
            col2im(col->devdata, nChannels, bottomHeight, bottomWidth,
                   kH, kW, dilH, dilW, padH, padW,
                   dH, dW, bottom->devdata + n * bottom_stride);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                PyErr_Format(PyExc_RuntimeError,
                             "GpuCorrMM encountered a CUDA error in col2im: %s\n"
                             "This could be a known bug in CUDA, please see the "
                             "GpuCorrMM() documentation.\n",
                             hipGetErrorString(err));
                Py_DECREF(col);
                return NULL;
            }
        }
    }
    // Free temporary columns
    Py_DECREF(col);

    return output;
}

CudaNdarray* corrMMWrapper(CudaNdarray *const bottom,
                    CudaNdarray *const weight,
                    CudaNdarray *const top,
                    int direction,
                    int dH, int dW,
                    int dilH, int dilW,
                    int padH, int padW,
                    const int callBinary = 0)
{
    if (callBinary == 1) {
        return corrMMBinary(bottom, weight, top, direction, dH, dW, dilH, dilW, padH, padW);
    } else {
        return corrMM(bottom, weight, top, direction, dH, dW, dilH, dilW, padH, padW);
    }
}

